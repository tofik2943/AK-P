#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h> 
#include "hip/hip_runtime.h"
#include ""



hipError_t addWithCuda(float *c, const float *tab, float alpha, int arraySize);		//NIE WIEM CZY POWINIEN BYC WSKAZNIK NA ALPHA/ARRAYSIZE


__global__ void addKernel(float *c, const float *tab, float alpha) {
	int i = threadIdx.x; //zaczyna sie od zera
	float x = tab[i * 3];
	float y = tab[i * 3 + 1];
	x = x * cos(alpha) - y * sin(alpha);
	y = x * sin(alpha) + y * cos(alpha);
	c[i * 3] = x;
	c[i * 3 + 1] = y;
}


/*
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
*/
int main()
{
	float alpha = 0;
	const int arraySize = 21; //7punkt�w po 3 wsp�rz�dne
	const float tab[arraySize] = { 1, 2, 3, 4, 5, 6 ,7, 1, 2, 3, 4, 5, 6 ,7,  1, 2, 3, 4, 5, 6 ,7 };

	float c[arraySize] = { 1, 2, 3, 4, 5, 6 ,7, 1, 2, 3, 4, 5, 6 ,7,  1, 2, 3, 4, 5, 6 ,7, };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, tab, alpha, arraySize);	// jakies wskazniki?
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	/*
	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]); */

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, float  *tab, float *alpha, int *arraySize)		//nie wiem dlaczego powinien byc wskaznik alpha
{
	float *dev_tab;
	float *dev_c;
	float *dev_alpha;
	const int dev_arraySize = 21;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&arraySize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, dev_arraySize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_tab, dev_arraySize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_alpha, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.

	/*

hipError_t hipMemcpy	(	void * 	dst,
const void * 	src,
size_t 	count,
enum hipMemcpyKind 	kind
)

	*/
	/*
	cudaStatus = hipMemcpy(dev_c, c, arraySize * sizeof(float), hipMemcpyHostToDevice);		//nie wiem czy �adowanie warto�ci poprzedniego elementu tablicy C ma sens
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	*/
	cudaStatus = hipMemcpy(dev_tab, tab, *arraySize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);		//(void**)&dev_alpha  ????
	//ale alpha to nazwa po stronie hosta
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}




	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, 7 >> > (c, tab, alpha);															//   <<<ile blok�w , ile w�tk�w w bloku>>> ?

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, dev_arraySize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_tab);
	hipFree(dev_alpha);
	//hipFree(dev_arraySize);
	return cudaStatus;
}
